#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2015, September 7 - October 6
// ###
// ###
// ### Thomas Moellenhoff, Robert Maier, Caner Hazirbas
// ###
// ###
// ###
// ### THIS FILE IS SUPPOSED TO REMAIN UNCHANGED
// ###
// ###

#include "aux.h"
#include <iostream>
#include <stdio.h>
using namespace std;

__global__ void init(float* xbar, float* xcur, float* xn, float* y1, float* y2, float* img, int w, int h, int nc) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    
    if (x < w && y < h) {
        int i;
        float val;

        for (int z = 0; z < nc; z++) {
            i = x + w * y + w * h * z;
            val = img[i];
            xbar[i] = val;
            xn[i] = val;
            xcur[i] = val;
            y1[i] = 0.f;
            y2[i] = 0.f;
        }
    }
}

__global__ void primal_descent(float* y1, float* y2, float* xbar, float sigma, float nu, float lambda, int w, int h, int nc) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    
    if (x < w && y < h) {
        int i;
        float val, x1, x2, norm;
        float factor = (2.0 * lambda) / (sigma + 2.f * lambda);
        float bound = sqrtf((nu / lambda) * sigma * (sigma + 2.f * lambda));

        for (int z = 0; z < nc; z++) {
            i = x + w * y + w * h * z;

            val = xbar[i];
            x1 = (x+1<w) ? (xbar[(x+1) + w * y + w * h * z] - val) : 0.f;
            x2 = (y+1<h) ? (xbar[x + w * (y+1) + w * h * z] - val) : 0.f;

            x1 = y1[i] + sigma * x1;
            x2 = y2[i] + sigma * x2;

            norm = sqrtf(x1*x1+x2*x2);

            y1[i] = norm * factor <= bound ? factor * x1 : 0.f;
            y2[i] = norm * factor <= bound ? factor * x2 : 0.f;
        }
    }
}

__global__ void dual_ascent(float* xn, float* xcur, float* y1, float* y2, float* img, float tau, int w, int h, int nc) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    
    if (x < w && y < h) {
        int i;
        float d1, d2, val;

        for (int z = 0; z < nc; z++) {
            i = x + w * y + w * h * z;

            d1 = y1[i] - (x>0 ? y1[(x-1) + w * y + w * h * z] : 0.f);
            d2 = y2[i] - (y>0 ? y2[x + w * (y-1) + w * h * z] : 0.f);
            val = xcur[i] + tau * (d1 + d2);
            xn[i] = (val + 2.f * tau * img[i]) / (1.f + 2.f * tau);
        }
    }
}

__global__ void extrapolate(float* xbar, float* xcur, float* xn, float theta, int w, int h, int nc) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    
    if (x < w && y < h) {
        int i;

        for (int z = 0; z < nc; z++) {
            i = x + w * y + w * h * z;

            xbar[i] = xn[i] + theta * (xn[i] - xcur[i]);
            xcur[i] = xn[i];
        }
    }
}

__global__ void solution(float* img, float* xbar, int w, int h, int nc) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x < w && y < h) {
        int i;

        for (int z = 0; z < nc; z++) {   
            i = x + w * y + w * h * z;

            img[i] = xbar[i];
        }
    }
}

int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;

    // Reading command line parameters:
    if (argc <= 2) { cout << "Usage: " << argv[0] << " -i <image> -o <output_image> [-repeats <repeats>] [-gray]" << endl; return 1; }
    
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    
    // output image
    string output = "";
    bool retO = getParam("o", output, argc, argv);
    if (!retO) cerr << "ERROR: no output image specified" << endl;

    // number of computation repetitions to get a better run time measurement
    int repeats = 140;
    getParam("repeats", repeats, argc, argv);
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);

    // load the input image as grayscale if "-gray" is specifed
    float tau = 0.25f;
    getParam("tau", tau, argc, argv);

    // load the input image as grayscale if "-gray" is specifed
    float sigma = 0.5f;
    getParam("sigma", sigma, argc, argv);

    // load the input image as grayscale if "-gray" is specifed
    float lambda = 2.f;
    getParam("lambda", lambda, argc, argv);

    // load the input image as grayscale if "-gray" is specifed
    float nu = 0.1f;
    getParam("nu", nu, argc, argv);

    // load the input image as grayscale if "-gray" is specifed
    float theta = 1.f;
    getParam("theta", theta, argc, argv);
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;

    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    int dim = w*h*nc;
    int nbyted = dim*sizeof(float);

    cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers

    // allocate raw input image array
    float* h_img  = new float[(size_t)dim];

    // allocate raw input image for GPU
    float* d_imgInOut; hipMalloc(&d_imgInOut, nbyted); CUDA_CHECK;

    float* d_x; hipMalloc(&d_x, nbyted); CUDA_CHECK;
    float* d_xbar; hipMalloc(&d_xbar, nbyted); CUDA_CHECK;
    float* d_xcur; hipMalloc(&d_xcur, nbyted); CUDA_CHECK;

    float* d_y1; hipMalloc(&d_y1, nbyted); CUDA_CHECK;
    float* d_y2; hipMalloc(&d_y2, nbyted); CUDA_CHECK;

    // Init raw input image array
    convert_mat_to_layered (h_img, mIn);

    // copy host memory
    hipMemcpy(d_imgInOut, h_img, nbyted, hipMemcpyHostToDevice); CUDA_CHECK;

    // launch kernel
    dim3 block = dim3(32, 8, 1);
    dim3 grid = dim3((w + block.x - 1) / block.x, (h + block.y - 1) / block.y, 1);

    Timer timer; timer.start();

    init <<<grid, block>>> (d_xbar, d_xcur, d_x, d_y1, d_y2, d_imgInOut, w, h, nc);
    for (int i = 1; i <= repeats; i++)
    {
        primal_descent <<<grid, block>>> (d_y1, d_y2, d_xbar, sigma, nu, lambda, w, h, nc);
        dual_ascent <<<grid, block>>> (d_x, d_xcur, d_y1, d_y2, d_imgInOut, tau, w, h, nc);
        theta = 1.f / sqrtf(1.f + 4.f * tau);
        tau *= theta;
        sigma /= theta;
        extrapolate <<<grid, block>>> (d_xbar, d_xcur, d_x, theta, w, h, nc);
    }
    solution <<<grid, block>>> (d_imgInOut, d_x, w, h, nc);

    timer.end();  float t = timer.get();  // elapsed time in seconds
    cout << "Time: " << t << " s" << endl;

    hipMemcpy(h_img, d_imgInOut, nbyted, hipMemcpyDeviceToHost); CUDA_CHECK;

    // show input image
    showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

    // show output image: first convert to interleaved opencv format from the layered raw array
    convert_layered_to_mat(mOut, h_img);
    showImage("Output", mOut, 100+w+40, 100);

    // wait for key inputs
    cv::waitKey(0);

    // save input and result
    cv::imwrite(output, mOut*255.f);

    // free GPU memory
    hipFree(d_imgInOut); CUDA_CHECK;
    
    hipFree(d_x); CUDA_CHECK;
    hipFree(d_xbar); CUDA_CHECK;
    hipFree(d_xcur); CUDA_CHECK;

    hipFree(d_y1); CUDA_CHECK;
    hipFree(d_y2); CUDA_CHECK;

    // free allocated arrays
    delete[] h_img;

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}