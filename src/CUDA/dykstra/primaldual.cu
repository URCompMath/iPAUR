#include "hip/hip_runtime.h"
// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2015, September 7 - October 6
// ###
// ###
// ### Thomas Moellenhoff, Robert Maier, Caner Hazirbas
// ###
// ###
// ###
// ### THIS FILE IS SUPPOSED TO REMAIN UNCHANGED
// ###
// ###

#include "aux.h"
#include <iostream>
#include <stdio.h>
using namespace std;

void parameterToFile(string filename,int repeats,bool gray,int level,float tau,float sigma,float lambda,float nu,int w,int h,int nc,size_t available,size_t total,float t,int iter) {
    FILE *file;
    file = fopen(filename.c_str(), "w");
    if(file == NULL)
        printf("ERROR: Could not open file!");
    else {
        fprintf(file, "image: %d x %d x %d\n", w, h, nc);
        fprintf(file,"repeats: %d\n", repeats);
        fprintf(file,"gray: %d\n", gray);
        fprintf(file,"level: %d\n", level);
        fprintf(file,"tau: %f\n", tau);
        fprintf(file,"sigma: %f\n", sigma);
        fprintf(file,"lambda: %f\n", lambda);
        fprintf(file,"nu: %f\n", nu);
        fprintf(file, "GPU Memory: %zd - %zd = %f GB\n", total, available, (total-available)/pow(10,9));
        fprintf(file, "time: %f s\n", t);
        fprintf(file, "iterations: %d\n", iter);
    }
    fclose (file);
}

void parameterToConsole(string filename,int repeats,bool gray,int level,float tau,float sigma,float lambda,float nu,int w,int h,int nc,size_t available,size_t total,float t,int iter) {
    printf( "image: %d x %d x %d\n", w, h, nc);
    printf("repeats: %d\n", repeats);
    printf("gray: %d\n", gray);
    printf("level: %d\n", level);
    printf("tau: %f\n", tau);
    printf("sigma: %f\n", sigma);
    printf("lambda: %f\n", lambda);
    printf("nu: %f\n", nu);
    printf( "GPU Memory: %zd - %zd = %f GB\n", total, available, (total-available)/pow(10,9));
    printf( "time: %f s\n", t);
    printf( "iterations: %d\n", iter);
}

__device__ float bound(float x1, float x2, float lambda, float k, float l, float f)
{
    return 0.25f * (x1*x1 + x2*x2) - lambda * pow(k / l - f, 2);
}

__device__ float interpolate(float k, float uk0, float uk1, float l)
{
    return (k + (0.5 - uk0) / (uk1 - uk0)) / l;
}

__device__ void on_parabola(float* u1, float* u2, float* u3, float x1, float x2, float x3, float f, float L, float lambda, float k, int j, float l)
{
    float y = x3 + lambda * pow(k / l - f, 2);
    float norm = sqrtf(x1*x1+x2*x2);
    float v = 0.f;
    float a = 2.f * 0.25f * norm;
    float b = 2.f / 3.f * (1.f - 2.f * 0.25f * y);
    float d = b < 0 ? (a - pow(sqrt(-b), 3)) * (a + pow(sqrt(-b), 3)) : a*a + b*b*b;
    float c = pow((a + sqrt(d)), 1.f/3.f);
    if (d >= 0) {
        v = c == 0 ? 0.f : c - b / c;
    } else {
        v = 2.f * sqrt(-b) * cos((1.f / 3.f) * acos(a / (pow(sqrt(-b), 3))));
    }
    u1[j] = norm == 0 ? 0.f : (v / (2.0 * 0.25f)) * x1 / norm;
    u2[j] = norm == 0 ? 0.f : (v / (2.0 * 0.25f)) * x2 / norm;
    u3[j] = bound(u1[j], u2[j], lambda, k, l, f);
}

__global__ void project_on_parabola(float* u1, float* u2, float* u3, float* v1, float* v2, float* v3, float* img, float L, float lambda, int k, int w, int h, int l)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    

    if (x < w && y < h && z < l)
    {
        int index = x + w * y;
        int i = x + w * y + w * h * z + (k-1) * w * h * l;
        int j = x + w * y + w * h * z + k * w * h * l;

        float f = img[index];
        float x1 = u1[i] - v1[j];
        float x2 = u2[i] - v2[j];
        float x3 = u3[i] - v3[j];
        float bound_val = bound(x1, x2, lambda, (z+1.f), l, f);

        if (x3 < bound_val) {
            on_parabola(u1, u2, u3, x1, x2, x3, f, L, lambda, (z+1.f), j, l);
        } else {
            u1[j] = x1;
            u2[j] = x2;
            u3[j] = x3;
        }
    }
}

__global__ void soft_shrinkage(float* u1, float* u2, float* u3, float* v1, float* v2, float* v3, float nu, int k1, int k2, int P, int w, int h, int l)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    const float K = (float)(k2 - k1 + 1);

    if (x < w && y < h)
    {
        int i, j;
        float s1 = 0.f;
        float s2 = 0.f;
        float s01 = 0.f;
        float s02 = 0.f;
        float x1 = 0.f;
        float x2 = 0.f;

        for (int k = k1; k <= k2; k++)
        {
            i = x + w * y + k * w * h + (P-1) * w * h * l;
            j = x + w * y + k * w * h + P * w * h * l;
            x1 = u1[i] - v1[j];
            x2 = u2[i] - v2[j];
            s01 += x1;
            s02 += x2;
        }

        float norm = sqrtf(s01*s01+s02*s02);

        s1 = norm <= nu ? s01 : (nu * s01 / norm);
        s2 = norm <= nu ? s02 : (nu * s02 / norm);

        for (int k = 0; k < l; k++)
        {
            i = x + w * y + k * w * h + (P-1) * w * h * l;
            j = x + w * y + k * w * h + P * w * h * l;
            x1 = u1[i] - v1[j];
            x2 = u2[i] - v2[j];
            if (k >= k1 && k <= k2) {
                u1[j] = x1 + (s1 - s01) / K;
                u2[j] = x2 + (s2 - s02) / K;
            } else {
                u1[j] = x1;
                u2[j] = x2;
            }
            u3[j] = u3[i] - v3[j];
        }
    }
}

__global__ void init(float* xbar, float* xcur, float* xn, float* y1, float* y2, float* y3, float* img, int w, int h, int l)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    
    if (x < w && y < h)
    {
        float img_val = img[x + w * y];
        for (int k = 0; k < l; k++)
        {
            int index = x + w * y + k * w * h;
            xn[index] = img_val;
            xcur[index] = img_val;
            xbar[index] = img_val;
            y1[index] = 0.f;
            y2[index] = 0.f;
            y3[index] = 0.f;
        }
    }
}

__global__ void isosurface(float* img, float* xbar, int w, int h, int l)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x < w && y < h)
    {
        float val = 0.f;
        float uk0 = 0.f;
        float uk1 = 0.f;

        for (int k = 0; k < l-1; k++)
        {
            uk0 = xbar[x + w * y + k * w * h];
            uk1 = xbar[x + w * y + (k+1) * w * h];
            if (uk0 > 0.5 && uk1 <= 0.5) {
                val = interpolate(k+1, uk0, uk1, l);
                break;
            } else {
                val = 1.f;
            }
        }
        
        img[x + w * y] = val;
    }
}

__global__ void set_y(float* y1, float* y2, float* y3, float* u1, float* u2, float* u3, int w, int h, int l, int p)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    

    if (x < w && y < h && z < l)
    {
        int i = x + w * y + w * h * z;
        int j = x + w * y + w * h * z + (p-1) * w * h * l;
        y1[i] = u1[j];
        y2[i] = u2[j];
        y3[i] = u3[j];
    }
}

__global__ void set_u_v(float* u1, float* u2, float* u3, float* v1, float* v2, float* v3, float* dx, float* dy, float* dz, int w, int h, int l, int p)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    
    if (x < w && y < h && z < l)
    {
        int i = x + w * y + w * h * z;
        int j;
        for (int k = 0; k < p; k++)
        {
            j = x + w * y + w * h * z + k * w * h * l;

            u1[j] = k < p-1 ? 0.f : dx[i];
            u2[j] = k < p-1 ? 0.f : dy[i];
            u3[j] = k < p-1 ? 0.f : dz[i];

            v1[j] = 0.f;
            v2[j] = 0.f;
            v3[j] = 0.f;
            
        }
    }
}

__global__ void update_v(float* v1, float* v2, float* v3, float* u1, float* u2, float* u3, int w, int h, int l, int k)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    

    if (x < w && y < h && z < l)
    {
        int i = x + w * y + w * h * z + k * w * h * l;
        int j = x + w * y + w * h * z + (k-1) * w * h * l;
        v1[i] = u1[i] - (u1[j] - v1[i]);
        v2[i] = u2[i] - (u2[j] - v2[i]);
        v3[i] = u3[i] - (u3[j] - v3[i]);
    }
}

__global__ void set_u_zero(float* u1, float* u2, float* u3, int w, int h, int l, int p)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    

    if (x < w && y < h && z < l)
    {
        int i = x + w * y + w * h * z;
        int j = x + w * y + w * h * z + (p-1) * w * h * l;
        u1[i] = u1[j];
        u2[i] = u2[j];
        u3[i] = u3[j];
    }
}

__global__ void gradient(float* dx, float* dy, float* dz, float* y1, float* y2, float* y3, float* xbar, float sigma, int w, int h, int l)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    
    if (x < w && y < h && z < l)
    {
        float val = xbar[x + w * y + w * h * z];
        float x1 = (x+1<w) ? (xbar[(x+1) + w * y + w * h * z] - val) : 0.f;
        float x2 = (y+1<h) ? (xbar[x + w * (y+1) + w * h * z] - val) : 0.f;
        float x3 = (z+1<l) ? (xbar[x + w * y + w * h * (z+1)] - val) : 0.f;
        dx[x + w * y + w * h * z] = y1[x + w * y + w * h * z] + sigma * x1;
        dy[x + w * y + w * h * z] = y2[x + w * y + w * h * z] + sigma * x2;
        dz[x + w * y + w * h * z] = y3[x + w * y + w * h * z] + sigma * x3;
    }
}

__global__ void clipping(float* xn, float* xcur, float* y1, float* y2, float* y3, float tau, int w, int h, int l)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    
    if (x < w && y < h && z < l)
    {
        float d1 = y1[x + w * y + w * h * z] - (x>0 ? y1[(x-1) + w * y + w * h * z] : 0.f);
        float d2 = y2[x + w * y + w * h * z] - (y>0 ? y2[x + w * (y-1) + w * h * z] : 0.f);
        float d3 = y3[x + w * y + w * h * z] - (z>0 ? y3[x + w * y + w * h * (z-1)] : 0.f);
        float val = xcur[x + w * y + w * h * z] + tau * (d1 + d2 + d3);
        if (z == 0) {
            xn[x + w * y + w * h * z] = 1.f;
        } else if (z == l-1) {
            xn[x + w * y + w * h * z] = 0.f;
        } else {
            xn[x + w * y + w * h * z] = fmin(1.f, fmax(0.f, val));
        }
    }
}

__global__ void extrapolate(float* xbar, float* xcur, float* xn, int w, int h, int l)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z = threadIdx.z + blockDim.z * blockIdx.z;
    
    int i = x + w * y + w * h * z;

    if (x < w && y < h && z < l) {
        xbar[i] = 2 * xn[i] - xcur[i];
        xcur[i] = xn[i];
    }
}

int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;

    // Reading command line parameters:

    if (argc <= 2) { cout << "Usage: " << argv[0] << " -i <image> -o <output_image> [-repeats <repeats>] [-gray]" << endl; return 1; }
    
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> -o <output_image> -data <data.txt> -parm <parameter.txt> [-repeats <repeats>] [-gray]" << endl; return 1; }

    // output image
    string output = "";
    bool retO = getParam("o", output, argc, argv);
    if (!retO) cerr << "ERROR: no output image specified" << endl;

    // parameter values
    string parm = "";
    bool ret2 = getParam("parm", parm, argc, argv);
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1000;
    getParam("repeats", repeats, argc, argv);

    // number of computation repetitions to get a better run time measurement
    int dykstra = 10;
    getParam("dykstra", dykstra, argc, argv);
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);

    // load the input image as grayscale if "-gray" is specifed
    int level = 16;
    getParam("level", level, argc, argv);

    // load the input image as grayscale if "-gray" is specifed
    float L = sqrtf(12);
    getParam("L", L, argc, argv);

    // load the input image as grayscale if "-gray" is specifed
    // float tau = 1000;
    float tau = 1.f/L;
    getParam("tau", tau, argc, argv);
    
    // load the input image as grayscale if "-gray" is specifed
    float sigma = 1.f/(L*L*tau);
    getParam("sigma", sigma, argc, argv);

    // load the input image as grayscale if "-gray" is specifed
    float lambda = 0.1;
    getParam("lambda", lambda, argc, argv);

    // load the input image as grayscale if "-gray" is specifed
    float nu = 0.01f;
    getParam("nu", nu, argc, argv);
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;

    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    int dim = w*h*nc;
    int size = w*h*nc*level;
    // int projections = level*(level-1)/2 + level + 2;
    int projections = level * (level+1) / 2 + 1 + 1;
    int nbytes = size*sizeof(float);
    int nbyted = dim*sizeof(float);
    int nbytep = projections*size*sizeof(float);

    cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers

    float nrj = 0.f;

    // allocate raw input image array
    // allocate raw input image array
    float* h_u = new float[(size_t)size];
    float* h_un = new float[(size_t)size];
    float* h_imgIn  = new float[(size_t)dim];
    float* h_imgOut = new float[(size_t)dim];

    // allocate raw input image for GPU
    float* d_imgInOut; hipMalloc(&d_imgInOut, nbyted); CUDA_CHECK;
    // float* d_imgOut;hipMalloc(&d_imgOut, nbyted); CUDA_CHECK;

    float* d_x; hipMalloc(&d_x, nbytes); CUDA_CHECK;
    float* d_xbar; hipMalloc(&d_xbar, nbytes); CUDA_CHECK;
    float* d_xcur; hipMalloc(&d_xcur, nbytes); CUDA_CHECK;

    float* d_delX; hipMalloc(&d_delX, nbytes); CUDA_CHECK;
    float* d_delY; hipMalloc(&d_delY, nbytes); CUDA_CHECK;
    float* d_delZ; hipMalloc(&d_delZ, nbytes); CUDA_CHECK;

    float* d_y1; hipMalloc(&d_y1, nbytes); CUDA_CHECK;
    float* d_y2; hipMalloc(&d_y2, nbytes); CUDA_CHECK;
    float* d_y3; hipMalloc(&d_y3, nbytes); CUDA_CHECK;

    float* d_u1; hipMalloc(&d_u1, nbytep); CUDA_CHECK;
    float* d_u2; hipMalloc(&d_u2, nbytep); CUDA_CHECK;
    float* d_u3; hipMalloc(&d_u3, nbytep); CUDA_CHECK;

    float* d_v1; hipMalloc(&d_v1, nbytep); CUDA_CHECK;
    float* d_v2; hipMalloc(&d_v2, nbytep); CUDA_CHECK;
    float* d_v3; hipMalloc(&d_v3, nbytep); CUDA_CHECK;

    size_t available, total;
    hipMemGetInfo(&available, &total);

    // alloc GPU memory

    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;

#endif

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered (h_imgIn, mIn);

    // copy host memory
    hipMemcpy(d_imgInOut, h_imgIn, nbyted, hipMemcpyHostToDevice); CUDA_CHECK;

    // launch kernel
    dim3 block = dim3(32, 8, 4);
    dim3 grid = dim3((w + block.x - 1) / block.x, (h + block.y - 1) / block.y, (level + block.z - 1) / block.z);
    dim3 block_iso = dim3(32, 8, 1);
    dim3 grid_iso = dim3((w + block_iso.x - 1) / block_iso.x, (h + block_iso.y - 1) / block_iso.y, 1);

    Timer timer; timer.start();

    int count_p = projections;
    int iter;

    init <<<grid_iso, block_iso>>> (d_xbar, d_xcur, d_x, d_y1, d_y2, d_y3, d_imgInOut, w, h, level);

    for (iter = 0; iter < repeats; iter++)
    {        
        gradient <<<grid, block>>> (d_delX, d_delY, d_delZ, d_y1, d_y2, d_y3, d_xbar, sigma, w, h, level);
        set_u_v <<<grid, block>>> (d_u1, d_u2, d_u3, d_v1, d_v2, d_v3, d_delX, d_delY, d_delZ, w, h, level, projections);
        for (int j = 0; j < dykstra; j++)
        {            
            count_p = 1;
            set_u_zero <<<grid, block>>> (d_u1, d_u2, d_u3, w, h, level, projections);
            project_on_parabola <<<grid, block>>> (d_u1, d_u2, d_u3, d_v1, d_v2, d_v3, d_imgInOut, L, lambda, count_p, w, h, level);
            update_v <<<grid, block>>> (d_v1, d_v2, d_v3, d_u1, d_u2, d_u3, w, h, level, count_p);
            count_p++;
            
            for (int k1 = 0; k1 < level; k1++)
            {
                for (int k2 = k1; k2 < level; k2++)
                {
                    soft_shrinkage <<<grid_iso, block_iso>>> (d_u1, d_u2, d_u3, d_v1, d_v2, d_v3, nu, k1, k2, count_p, w, h, level);
                    update_v <<<grid, block>>> (d_v1, d_v2, d_v3, d_u1, d_u2, d_u3, w, h, level, count_p);
                    count_p++;
                }
            }
        }
        set_y <<<grid, block>>> (d_y1, d_y2, d_y3, d_u1, d_u2, d_u3, w, h, level, projections);
        clipping <<<grid, block>>> (d_x, d_xcur, d_y1, d_y2, d_y3, tau, w, h, level);
        hipMemcpy(h_u, d_x, nbytes, hipMemcpyDeviceToHost); CUDA_CHECK;
        hipMemcpy(h_un, d_xcur, nbytes, hipMemcpyDeviceToHost); CUDA_CHECK;
        nrj = 0.f;
        for (int i = 0; i < size; i++)
        {
            nrj += fabs(h_u[i] - h_un[i]);
        }
        if (nrj/(w*h*level) <= 5*1E-5) break;
        extrapolate <<<grid, block>>> (d_xbar, d_xcur, d_x, w, h, level);
    }
    isosurface <<<grid_iso, block_iso>>> (d_imgInOut, d_x, w, h, level);
    
    timer.end();  float t = timer.get();  // elapsed time in seconds

    hipMemcpy(h_imgOut, d_imgInOut, nbyted, hipMemcpyDeviceToHost); CUDA_CHECK;
    // dualEnergy(data, h_energy, count);
    if (!ret2) {
        parameterToConsole(parm,repeats,gray,level,tau,sigma,lambda,nu,w,h,nc,available,total,t,iter);
    } else {
        parameterToFile(parm,repeats,gray,level,tau,sigma,lambda,nu,w,h,nc,available,total,t,iter);
    }

    // free GPU memory
    hipFree(d_imgInOut); CUDA_CHECK;
    // hipFree(d_imgInOut); CUDA_CHECK;
    
    hipFree(d_x); CUDA_CHECK;
    hipFree(d_xbar); CUDA_CHECK;
    hipFree(d_xcur); CUDA_CHECK;

    hipFree(d_delX); CUDA_CHECK;
    hipFree(d_delY); CUDA_CHECK;
    hipFree(d_delZ); CUDA_CHECK;

    hipFree(d_y1); CUDA_CHECK;
    hipFree(d_y2); CUDA_CHECK;
    hipFree(d_y3); CUDA_CHECK;

    hipFree(d_u1); CUDA_CHECK;
    hipFree(d_u2); CUDA_CHECK;
    hipFree(d_u3); CUDA_CHECK;

    hipFree(d_v1); CUDA_CHECK;
    hipFree(d_v2); CUDA_CHECK;
    hipFree(d_v3); CUDA_CHECK;

    // show input image
    // showImage("Input", mIn, 100, 100);  // show at position (x_from_left=100,y_from_above=100)

    // show output image: first convert to interleaved opencv format from the layered raw array
    convert_layered_to_mat(mOut, h_imgOut);
    // showImage("Output", mOut, 100+w+40, 100);

    // ### Display your own output images here as needed

#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    // cv::waitKey(0);
#endif

    // save input and result
    // cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    cv::imwrite(output, mOut*255.f);

    // free allocated arrays
    delete[] h_u;
    delete[] h_un;
    delete[] h_imgIn;
    delete[] h_imgOut;
    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}
